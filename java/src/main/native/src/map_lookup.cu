#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

namespace cudf {
namespace {

void __device__ search_each_list(size_type list_index,
                        column_device_view input,
                        mutable_column_device_view output,
                        string_scalar_device_view lookup_key)
{
  if (input.is_null(list_index)) { // List row is null.
    output.element<size_type>(list_index) = -1;  // Not found.
    return;
  }

  auto offsets{input.child(0)};
  auto start_index{offsets.element<size_type>(list_index)};
  auto end_index{offsets.element<size_type>(list_index + 1)};

  auto key_column{input.child(1).child(0)};

  for (size_type list_element_index{start_index}; list_element_index < end_index; ++list_element_index) {
    if (!key_column.is_null(list_element_index) && key_column.element<string_view>(list_element_index) == lookup_key.value()) {
      output.element<size_type>(list_index) = list_element_index;
      return;
    }
  }

  output.element<size_type>(list_index) = -1;  // Not found.
}

template <int block_size>
__launch_bounds__(block_size) __global__
  void gpu_find_first(column_device_view input,
                      mutable_column_device_view output,
                      string_scalar_device_view lookup_key,
                      size_type* __restrict__ output_valid_count)
{
  size_type i      = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;
  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());

  while (i < input.size()) {
    volatile bool output_valid{true};
    search_each_list(i, input, output, lookup_key);

    // Set validity mask.
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_valid)};

    // Write out the mask.
    if (0 == threadIdx.x % cudf::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // Sum up valid_count across the whole block
  size_type block_valid_count =
    cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
}

size_type kernel_launcher(column_view const& input,
                          mutable_column_view const& output,
                          string_scalar& lookup_key,
                          hipStream_t stream)
{
  constexpr size_type block_size{256};
  cudf::detail::grid_1d grid{input.size(), block_size};

  auto input_device_view = cudf::column_device_view::create(input, stream);
  auto output_view       = mutable_column_device_view::create(output, stream);
  auto lookup_key_device_view{get_scalar_device_view(lookup_key)};
  rmm::device_scalar<size_type> device_valid_count{0, stream};

  gpu_find_first<block_size><<<grid.num_blocks, block_size, 0, stream>>>(
    *input_device_view, *output_view, lookup_key_device_view, device_valid_count.data());

  size_type valid_count = device_valid_count.value(stream);

  CHECK_CUDA(stream);

  return valid_count;
}
}  // namespace

namespace jni {
std::unique_ptr<column> map_lookup(column_view const& map_column,
                                   string_scalar lookup_key,
                                   rmm::mr::device_memory_resource* mr,
                                   hipStream_t stream)
{
  // Defensive checks.
  CUDF_EXPECTS(map_column.type().id() == type_id::LIST, "Expected LIST<STRUCT<key,value>>.");

  lists_column_view lcv{map_column};
  auto structs_column = lcv.get_sliced_child(stream);

  CUDF_EXPECTS(structs_column.type().id() == type_id::STRUCT, "Expected LIST<STRUCT<key,value>>.");

  structs_column_view scv{structs_column};
  CUDF_EXPECTS(structs_column.num_children() == 2, "Expected LIST<STRUCT<key,value>>.");
  CUDF_EXPECTS(structs_column.child(0).type().id() == type_id::STRING,
               "Expected LIST<STRUCT<key,value>>.");
  CUDF_EXPECTS(structs_column.child(1).type().id() == type_id::STRING,
               "Expected LIST<STRUCT<key,value>>.");

  // Two-pass plan: construct gather map, and then gather() on structs_column.child(1). Plan A.
  // Can do in one pass perhaps, but that's Plan B.

  auto gather_map              = make_numeric_column(data_type{cudf::type_to_id<size_type>()},
                                        map_column.size(),
                                        mask_state::UNINITIALIZED,
                                        stream,
                                        mr);
  auto gather_map_mutable_view = gather_map->mutable_view();

  auto valid_count = kernel_launcher(map_column, gather_map_mutable_view, lookup_key, stream);

  // Gather map is now available.

  auto values_column    = structs_column.child(1);
  auto table_for_gather = table_view{std::vector<cudf::column_view>{values_column}};

  auto gathered_table = cudf::detail::gather(table_for_gather,
                                             gather_map->view(),
                                             detail::out_of_bounds_policy::IGNORE,
                                             detail::negative_index_policy::NOT_ALLOWED,
                                             mr,
                                             stream);

  return std::make_unique<cudf::column>(std::move(gathered_table->get_column(0)));
}
} // namespace jni;
} // namespace cudf;